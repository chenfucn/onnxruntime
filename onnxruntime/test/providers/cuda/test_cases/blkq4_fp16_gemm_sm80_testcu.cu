#include "hip/hip_runtime.h"
/**
 * Copyright (c) Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License.
 *
 * Module Name:
 *    blkq4_fp16_gemm_sm80_testcu.cu
 *
 * Abstract:
 *   Test code for invoking block-wise quantized 4b GEMM kernels.
 *   This part requires CUTLASS header files, which do not play
 *   well with gtest headers.
 */

#include "core/mickey/blk_q4/f16_gemm_sm80.h"

#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"

#include "core/common/common.h"
#include "core/framework/float16.h"
#include "core/util/matrix_layout.h"

#include "blk_q4/f16_prepack_sm80.h"
#include "blkq4_fp16_gemm_sm80.h"

namespace onnxruntime {
namespace cuda{
namespace test{

Status sm80_supported(){
  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::ostringstream ss;
    ss << "Unable to obtain GPU device properties: " << hipGetErrorString(error);
    return Status(common::ONNXRUNTIME, common::ENGINE_ERROR, ss.str());
  }

  if (!((props.major * 10 + props.minor) >= 80)) {
    std::ostringstream ss;
    ss << "Device compute capability mismatch, desired 8.0, actual " << props.major << "." << props.minor;
    return Status(common::ONNXRUNTIME, common::ENGINE_ERROR, ss.str());
  }
  return Status::OK();
}

/**
 * @brief Reference implementation of GEMM
 *        Copied directly from cutlass util/reference/device/gemm.h
 *        for the strange reason that compiler insists on asking
 *        for explicit stream argument in kernel launch.
*/
template <
  typename ElementA,
  typename LayoutA,
  typename ElementB,
  typename LayoutB,
  typename ElementC,
  typename LayoutC,
  typename ScalarType,
  typename AccumulatorType
>
void compute_gemm_ref(
  cutlass::gemm::GemmCoord problem_size,
  ScalarType alpha,
  cutlass::TensorRef<ElementA, LayoutA> tensor_a,
  cutlass::TensorRef<ElementB, LayoutB> tensor_b,
  ScalarType beta,
  cutlass::TensorRef<ElementC, LayoutC> tensor_d,
  AccumulatorType initial_accum = AccumulatorType(0)) {

  // Blocking structure potentially improves performance of reference implementation
  // with a minor increase in complexity.
  //
  // Note, this reference implementation is NOT expected to approach peak performance.
  using OutputTile = cutlass::MatrixShape<4, 4>;

  dim3 block(16, 8);

  dim3 grid(
    (problem_size.m() + block.x * OutputTile::kRow - 1) / (block.x * OutputTile::kRow),
    (problem_size.n() + block.y * OutputTile::kColumn - 1) / (block.y * OutputTile::kColumn)
  );

  // Launch a GEMM kernel
  cutlass::reference::device::kernel::Gemm<
    cutlass::TensorRef<ElementA, LayoutA>,
    cutlass::TensorRef<ElementB, LayoutB>,
    cutlass::TensorRef<ElementC, LayoutC>,
    ScalarType,
    AccumulatorType,
    OutputTile,
    cutlass::multiply_add<AccumulatorType>,
    cutlass::NumericConverter<ElementC, ScalarType>
  ><<<grid, block, 0, 0>>>(
    problem_size,
    alpha,
    tensor_a,
    tensor_b,
    beta,
    tensor_d,
    tensor_d,
    initial_accum
  );
}
////////////////////////////////////////////////////////////////////////////////////////////////////

//
// Converting cutlass tensor to MatrixRef
//

template <
  typename Element,
  typename LayoutCutlass,
  typename Layout = std::conditional_t<std::is_same<LayoutCutlass, cutlass::layout::ColumnMajor>::value, ColumnMajorLayout, RowMajorLayout>
  >
__forceinline__
MatrixRef<Element, Layout, true> make_MatrixRef(cutlass::HostTensor<Element, LayoutCutlass> const& tensor) {
  static_assert(std::is_same<LayoutCutlass, cutlass::layout::ColumnMajor>::value
                || std::is_same<LayoutCutlass, cutlass::layout::RowMajor>::value);
  auto shape = make_Position(tensor.extent().row(), tensor.extent().column());
  auto* ptr = const_cast<typename std::remove_const<Element>::type *>(tensor.host_data());
  return MatrixRef<Element, Layout, true>(ptr, tensor.capacity(), shape);
}

template <
  typename Element,
  typename LayoutCutlass,
  typename Layout = std::conditional_t<std::is_same<LayoutCutlass, cutlass::layout::ColumnMajor>::value, ColumnMajorLayout, RowMajorLayout>
  >
__forceinline__
MatrixRef<Element const, Layout, true> make_ConstMatrixRef(cutlass::HostTensor<Element, LayoutCutlass> const& tensor) {
  static_assert(std::is_same<LayoutCutlass, cutlass::layout::ColumnMajor>::value
                || std::is_same<LayoutCutlass, cutlass::layout::RowMajor>::value);
  auto shape = make_Position(tensor.extent().row(), tensor.extent().column());
  return MatrixRef<Element const, Layout, true>(tensor.host_data(), tensor.capacity(), shape);
}

class SyncBuffer {
  public:
    SyncBuffer() {
    }

    ~SyncBuffer() {
      if (gpu_buffer_ != nullptr) {
        hipError_t error = hipFree(gpu_buffer_);
        // ORT_ENFORCE(error == hipSuccess, "Failed to free sync buffer: ", hipGetErrorString(error));
      }
    }

    void AllocCpuPtr(size_t byte_size) {
      ORT_ENFORCE(byte_size_ == 0, "Double allocation not allowed.");
      cpu_buffer_.resize(byte_size);
      byte_size_ = byte_size;
      hipError_t error = hipMalloc(&gpu_buffer_, byte_size_);
      ORT_ENFORCE(error == hipSuccess, "Failed to allocate sync buffer: ", hipGetErrorString(error));
    }


    uint8_t* CpuPtr() {
      return cpu_buffer_.data();
    }

    uint8_t* GpuPtr() {
      return reinterpret_cast<uint8_t*>(gpu_buffer_);
    }

    size_t Size() {
      return byte_size_;
    }

    void CopyToGpu() {
      hipError_t error = hipMemcpy(gpu_buffer_, cpu_buffer_.data(), byte_size_, hipMemcpyHostToDevice);
      // ORT_ENFORCE(error == hipSuccess, "Failed to copy sync buffer to GPU: ", hipGetErrorString(error));
    }

  private:
    std::vector<uint8_t> cpu_buffer_;
    size_t byte_size_{0};
    void* gpu_buffer_{nullptr};
};

template<int BlkSize, bool ColumnWiseQuantBlk, bool has_offsets>
class PackedBuf {

  public:
    PackedBuf(int rows, int cols) :
      rows_(rows), cols_(cols),
      packed_b_shape_(BlockwiseQuantization<MLFloat16, BlkSize, 4, ColumnWiseQuantBlk>::get_quant_weights_shape(rows, cols)),
      packed_meta_shape_(BlockwiseQuantization<MLFloat16, BlkSize, 4, ColumnWiseQuantBlk>::get_quant_meta_shape(rows, cols)) {
      size_t packed_buf_size = packed_b_shape_.product() + packed_meta_shape_.product() * sizeof(MLFloat16);
      if (has_offsets) {
        packed_buf_size += packed_meta_shape_.product();
      }

      packed_buf_.AllocCpuPtr(packed_buf_size);
    }

    ~PackedBuf() {}

    gsl::span<uint8_t> GetPackedWeights() {
      return gsl::make_span<uint8_t>(reinterpret_cast<uint8_t*>(packed_buf_.CpuPtr()),
                                    static_cast<size_t>(packed_b_shape_.product()));
    }

    gsl::span<MLFloat16> GetPackedScales() {
      auto* start = reinterpret_cast<MLFloat16*>(packed_buf_.CpuPtr() + packed_b_shape_.product());
      return gsl::make_span<MLFloat16>(start, static_cast<size_t>(packed_meta_shape_.product()));
    }

    gsl::span<uint8_t> GetPackedOffsets() {
      auto* start = reinterpret_cast<uint8_t*>(packed_buf_.CpuPtr() + packed_b_shape_.product()
          + packed_meta_shape_.product() * sizeof(half));
      return gsl::make_span<uint8_t>(start, static_cast<size_t>(packed_meta_shape_.product()));
    }

    gsl::span<uint8_t const> GetPackedWeightsGpu() {
      return gsl::make_span<uint8_t const>(reinterpret_cast<uint8_t const*>(packed_buf_.GpuPtr()),
                                          static_cast<size_t>(packed_b_shape_.product()));
    }

    gsl::span<half const> GetPackedScalesGpu() {
      half const* start = reinterpret_cast<half const*>(packed_buf_.GpuPtr() + packed_b_shape_.product());
      return gsl::make_span<half const>(start, static_cast<size_t>(packed_meta_shape_.product()));
    }

    gsl::span<uint8_t const> GetPackedOffsetsGpu() {
      uint8_t const* start = reinterpret_cast<uint8_t const*>(packed_buf_.GpuPtr() + packed_b_shape_.product()
          + packed_meta_shape_.product() * sizeof(half));
      return gsl::make_span<uint8_t const>(start, static_cast<size_t>(packed_meta_shape_.product()));
    }

    void pack_weights(gsl::span<uint8_t const> weights){
      switch (BlkSize)
      {
      case 16:
        if (ColumnWiseQuantBlk) {
          BlockwiseQuantization<MLFloat16, 16, 4, true>::prepack_weights(
              rows_, cols_, weights, GetPackedWeights());
        } else {
          BlockwiseQuantization<MLFloat16, 16, 4, false>::prepack_weights(
              rows_, cols_, weights, GetPackedWeights());
        }
        break;
      case 32:
        if (ColumnWiseQuantBlk) {
          BlockwiseQuantization<MLFloat16, 32, 4, true>::prepack_weights(
              rows_, cols_, weights, GetPackedWeights());
        } else {
          BlockwiseQuantization<MLFloat16, 32, 4, false>::prepack_weights(
              rows_, cols_, weights, GetPackedWeights());
        }
        break;
      case 64:
        if (ColumnWiseQuantBlk) {
          BlockwiseQuantization<MLFloat16, 64, 4, true>::prepack_weights(
              rows_, cols_, weights, GetPackedWeights());
        } else {
          BlockwiseQuantization<MLFloat16, 64, 4, false>::prepack_weights(
              rows_, cols_, weights, GetPackedWeights());
        }
        break;
      default:
        ORT_THROW("Unsupported block size: ", BlkSize);
      }
    }

    void pack_scales(gsl::span<MLFloat16 const> scales){
      switch (BlkSize)
      {
      case 16:
        if (ColumnWiseQuantBlk) {
          BlockwiseQuantization<MLFloat16, 16, 4, true>::prepack_quant_scales(
              rows_, cols_, scales, GetPackedScales());
        } else {
          BlockwiseQuantization<MLFloat16, 16, 4, false>::prepack_quant_scales(
              rows_, cols_, scales, GetPackedScales());
        }
        break;
      case 32:
        if (ColumnWiseQuantBlk) {
          BlockwiseQuantization<MLFloat16, 32, 4, true>::prepack_quant_scales(
              rows_, cols_, scales, GetPackedScales());
        } else {
          BlockwiseQuantization<MLFloat16, 32, 4, false>::prepack_quant_scales(
              rows_, cols_, scales, GetPackedScales());
        }
        break;
      case 64:
        if (ColumnWiseQuantBlk) {
          BlockwiseQuantization<MLFloat16, 64, 4, true>::prepack_quant_scales(
              rows_, cols_, scales, GetPackedScales());
        } else {
          BlockwiseQuantization<MLFloat16, 64, 4, false>::prepack_quant_scales(
              rows_, cols_, scales, GetPackedScales());
        }
        break;
      default:
        ORT_THROW("Unsupported block size: ", BlkSize);
      }
    }

    void pack_offsets(gsl::span<uint8_t const> offsets) {
      switch (BlkSize)
      {
      case 16:
        if (ColumnWiseQuantBlk) {
          BlockwiseQuantization<MLFloat16, 16, 4, true>::prepack_quant_offsets(
              rows_, cols_, offsets, GetPackedOffsets());
        } else {
          BlockwiseQuantization<MLFloat16, 16, 4, false>::prepack_quant_offsets(
              rows_, cols_, offsets, GetPackedOffsets());
        }
        break;
      case 32:
        if (ColumnWiseQuantBlk) {
          BlockwiseQuantization<MLFloat16, 32, 4, true>::prepack_quant_offsets(
              rows_, cols_, offsets, GetPackedOffsets());
        } else {
          BlockwiseQuantization<MLFloat16, 32, 4, false>::prepack_quant_offsets(
              rows_, cols_, offsets, GetPackedOffsets());
        }
        break;
      case 64:
        if (ColumnWiseQuantBlk) {
          BlockwiseQuantization<MLFloat16, 64, 4, true>::prepack_quant_offsets(
              rows_, cols_, offsets, GetPackedOffsets());
        } else {
          BlockwiseQuantization<MLFloat16, 64, 4, false>::prepack_quant_offsets(
              rows_, cols_, offsets, GetPackedOffsets());
        }
        break;
      default:
        ORT_THROW("Unsupported block size: ", BlkSize);
      }
    }

    void copy_to_gpu() {
      packed_buf_.CopyToGpu();
    }

  private:
    int rows_, cols_;
    Position<2> packed_b_shape_;
    Position<2> packed_meta_shape_;
    SyncBuffer packed_buf_;
};

/**
 * @brief Helper function to run the GEMM kernel for 4bits quantized gemm on SM80.
 * Only support fp16 for now.
*/
template<
    int block_size,
    bool column_wise_blocking,
    bool small_m,
    bool has_offsets>
Status blkq4_gemm_sm80(int m, int n, int k, hipStream_t stream,
                     gsl::span<half const> a,
                     gsl::span<uint8_t const> weights,
                     gsl::span<half const> scales,
                     gsl::span<uint8_t const> offsets,
                     gsl::span<half> output) {

  using ElementDequant = cutlass::half_t;
  using QuantBlocking =
    typename std::conditional<column_wise_blocking,
                     cutlass::MatrixShape<block_size, 1>,
                     cutlass::MatrixShape<1, block_size>>::type;

  using GemmRunner = BlkQ4F16GemmImpl<ElementDequant, QuantBlocking, small_m, has_offsets>;

  using ElementAccumulator = typename GemmRunner::ElementAccumulator;
  using ElementComputeEpilogue = typename GemmRunner::ElementComputeEpilogue;
  using ElementOutput = typename GemmRunner::ElementOutput;
  using ElementW = typename GemmRunner::ElementW;
  using ElementWPack = typename GemmRunner::ElementWPack;
  using ElementQScale = typename GemmRunner::ElementQScale;
  using ElementQOffset = typename GemmRunner::ElementQOffset;

  using LayoutInputA = typename GemmRunner::LayoutInputA;
  using LayoutOutput = typename GemmRunner::LayoutOutput;
  using LayoutInputWPack = typename GemmRunner::LayoutInputWPack;
  using LayoutInputQScale = typename GemmRunner::LayoutInputQScale;

  const cutlass::gemm::GemmCoord problem_size = {m, n, k};

  ORT_RETURN_IF_NOT(a.size_bytes() == m * k * sizeof(ElementDequant), "Activation tensor size is not correct");
  cutlass::TensorRef<ElementDequant const, LayoutInputA> ref_a(
    reinterpret_cast<ElementDequant const *>(a.data()),
    LayoutInputA::packed({m, k}));

  ORT_RETURN_IF_NOT(weights.size_bytes() == k/2 * n/2 * sizeof(ElementWPack), "weights size is not correct");
  cutlass::TensorRef<ElementWPack const, LayoutInputWPack> ref_W(
    reinterpret_cast<ElementWPack const *>(weights.data()),
    LayoutInputWPack::packed({k/2, n/2}));

  ORT_RETURN_IF_NOT(scales.size_bytes() == (k/QuantBlocking::kRow) * (n/QuantBlocking::kColumn) * sizeof(ElementQScale),
              "scales size is not correct");
  cutlass::TensorRef<ElementQScale const, LayoutInputQScale> ref_scales(
    reinterpret_cast<ElementQScale const *>(scales.data()),
    LayoutInputQScale::packed({k/QuantBlocking::kRow, n/QuantBlocking::kColumn}));

  ORT_RETURN_IF_NOT(output.size_bytes() == m * n * sizeof(ElementOutput), "output size is not correct");
  cutlass::TensorRef<ElementOutput, LayoutOutput> ref_output(
    reinterpret_cast<ElementOutput *>(output.data()),
    LayoutOutput::packed({m, n}));

  // run GEMM
  cutlass::Status status;
  if constexpr (has_offsets) {
    ORT_RETURN_IF_NOT(offsets.size_bytes() == (k/QuantBlocking::kRow) * (n/QuantBlocking::kColumn) * sizeof(ElementQOffset),
                "offsets size is not correct");
    cutlass::TensorRef<ElementQOffset const, LayoutInputQScale> ref_offsets(
      reinterpret_cast<ElementQOffset const *>(offsets.data()),
      LayoutInputQScale::packed({k/QuantBlocking::kRow, n/QuantBlocking::kColumn}));
    status = GemmRunner::run(
      stream, problem_size, ref_a, ref_W, ref_scales, ref_offsets,
      ref_output, ref_output);
  } else {
    status = GemmRunner::run(
      stream, problem_size, ref_a, ref_W, ref_scales,
      ref_output, ref_output);
  }
  ORT_RETURN_IF_NOT(status == cutlass::Status::kSuccess, "Kernel execution failed: ", cutlassGetStatusString(status));
  return Status::OK();
}

Status blkq4_fp16_gemm_sm80_dispatch(
  int block_size,
  bool column_wise_blocking,
  int m, int n, int k, hipStream_t stream,
  gsl::span<half const> a,
  gsl::span<uint8_t const> weights,
  gsl::span<half const> scales,
  gsl::span<uint8_t const> offsets,
  gsl::span<half> output) {

  switch (block_size)
  {
  case 16:
    if (column_wise_blocking) {
      if (m > 16) {
        if (offsets.empty())
          return blkq4_gemm_sm80<16, true, false, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<16, true, false, true>(m, n, k, stream, a, weights, scales, offsets, output);
      } else {
        if (offsets.empty())
          return blkq4_gemm_sm80<16, true, true, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<16, true, true, true>(m, n, k, stream, a, weights, scales, offsets, output);
      }
    } else {
      if (m > 16) {
        if (offsets.empty())
          return blkq4_gemm_sm80<16, false, false, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<16, false, false, true>(m, n, k, stream, a, weights, scales, offsets, output);
      } else {
        if (offsets.empty())
          return blkq4_gemm_sm80<16, false, true, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<16, false, true, true>(m, n, k, stream, a, weights, scales, offsets, output);
      }
    }
    break;

  case 32:
    if (column_wise_blocking) {
      if (m > 16) {
        if (offsets.empty())
          return blkq4_gemm_sm80<32, true, false, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<32, true, false, true>(m, n, k, stream, a, weights, scales, offsets, output);
      } else {
        if (offsets.empty())
          return blkq4_gemm_sm80<32, true, true, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<32, true, true, true>(m, n, k, stream, a, weights, scales, offsets, output);
      }
    } else {
      if (m > 16) {
        if (offsets.empty())
          return blkq4_gemm_sm80<32, false, false, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<32, false, false, true>(m, n, k, stream, a, weights, scales, offsets, output);
      } else {
        if (offsets.empty())
          return blkq4_gemm_sm80<32, false, true, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<32, false, true, true>(m, n, k, stream, a, weights, scales, offsets, output);
      }
    }
    break;

  case 64:
    if (column_wise_blocking) {
      if (m > 16) {
        if (offsets.empty())
          return blkq4_gemm_sm80<64, true, false, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<64, true, false, true>(m, n, k, stream, a, weights, scales, offsets, output);
      } else {
        if (offsets.empty())
          return blkq4_gemm_sm80<64, true, true, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<64, true, true, true>(m, n, k, stream, a, weights, scales, offsets, output);
      }
    } else {
      if (m > 16) {
        if (offsets.empty())
          return blkq4_gemm_sm80<64, false, false, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<64, false, false, true>(m, n, k, stream, a, weights, scales, offsets, output);
      } else {
        if (offsets.empty())
          return blkq4_gemm_sm80<64, false, true, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<64, false, true, true>(m, n, k, stream, a, weights, scales, offsets, output);
      }
    }
    break;
  }

  return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Unsupported block size: ", block_size);
}


//
// Invoking the kernel
//

template<
    int block_size,
    bool column_wise_blocking,
    bool small_m,
    bool has_offsets>
void run_blkq4_gemm(int m, int n, int k) {

  using ElementDequant = cutlass::half_t;
  using QuantBlocking =
    typename std::conditional<column_wise_blocking,
                     cutlass::MatrixShape<block_size, 1>,
                     cutlass::MatrixShape<1, block_size>>::type;

  using GemmRunner = BlkQ4F16GemmImpl<ElementDequant, QuantBlocking, small_m, has_offsets>;

  using ElementAccumulator = typename GemmRunner::ElementAccumulator;
  using ElementComputeEpilogue = typename GemmRunner::ElementComputeEpilogue;
  using ElementInputA = typename GemmRunner::ElementInputA;
  using ElementOutput = typename GemmRunner::ElementOutput;
  using ElementW = typename GemmRunner::ElementW;
  using ElementWPack = typename GemmRunner::ElementWPack;
  using ElementQScale = typename GemmRunner::ElementQScale;
  using ElementQOffset = typename GemmRunner::ElementQOffset;

  using LayoutInputA = typename GemmRunner::LayoutInputA;
  using LayoutOutput = typename GemmRunner::LayoutOutput;
  using LayoutInputWPack = typename GemmRunner::LayoutInputWPack;
  using LayoutInputQScale = typename GemmRunner::LayoutInputQScale;

  const cutlass::gemm::GemmCoord problem_size = {m, n, k};

  // Initialize tensors using CUTLASS helper functions
  cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(
      problem_size.mk());  // <- Create matrix A with dimensions M x K

  // Create weight matrix with dimensions K x N.
  // Actual weight type is int4, we use ElementW = uint8 to avoid possible compilation
  // troubles. Since the layout is column major, we are packing 2 weights in a column
  // into one int8
  cutlass::HostTensor<ElementW, LayoutInputWPack> tensor_weight(
      {problem_size.k()/2, problem_size.n()});

  // Create weight quantization scale and offset with dimensions K x N
  cutlass::HostTensor<ElementQScale, LayoutInputQScale> tensor_scale(
      {problem_size.k()/QuantBlocking::kRow, problem_size.n()/QuantBlocking::kColumn});
  cutlass::HostTensor<ElementQScale, cutlass::layout::ColumnMajor> tensor_scale1(
      {problem_size.k()/QuantBlocking::kRow, problem_size.n()/QuantBlocking::kColumn});

  cutlass::HostTensor<ElementQOffset, LayoutInputQScale> tensor_offset(
      {problem_size.k()/QuantBlocking::kRow, problem_size.n()/QuantBlocking::kColumn});
  cutlass::HostTensor<ElementQOffset, cutlass::layout::ColumnMajor> tensor_offset1(
      {((problem_size.k()/QuantBlocking::kRow) + 1) / 2, problem_size.n()/QuantBlocking::kColumn});

  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_d(
      problem_size.mn());  // <- Create matrix D with dimensions M x N used to store output from
                           // CUTLASS kernel

  // Fill input and output matrices on host using CUTLASS helper functions
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_a.host_view(),
      1,
      ElementInputA(4),
      ElementInputA(-4),
      2);  // <- Fill matrix A on host with uniform-distribution random data
  cutlass::reference::host::TensorFill(
      tensor_d.host_view());  // <- fill matrix D on host with zeros

  //
  // For testing quantization and dequantization, it is not straight
  // forward to avoid flaky tests due to rounding errors. The way we
  // try to achieve this is to:
  // 1. Generate a set of quantized weights, scales and offsets
  // 2. Dequantize the weights
  // 3. Quantize the dequantized weights
  // 4. Compare the dequantied-and-then-quantized weights with
  //    the original quantized weights
  //
  // Random filling of the initial values are key to get this right.
  // For weights, we must ensure each block gets a full range of
  // values, i.e. must contain 0 and 15. And for scales, they must
  // all be positive.
  //

  int v = 7;
  for (int c = 0; c < tensor_weight.extent()[1]; c++) {
    for (int r = 0; r < tensor_weight.extent()[0]; ++r) {
      uint8_t v0 = static_cast<uint8_t>(v);
      v = (v + 5) % 16;
      if (v == 11 || v == 7 || v == 3) {
        // making the cycle 13 instead of 16, avoiding same values in a row
        v = (v + 5) % 16;
      }
      uint8_t v1 = 0;
      v1 = static_cast<uint8_t>(v);
      v = (v + 5) % 16;
      if (v == 11 || v == 7 || v == 3) {
        // making the cycle 13 instead of 16, avoiding same values in a row
        v = (v + 5) % 16;
      }

      tensor_weight.at({r, c}) = ElementW((v1 << 4) | v0);
    }
  }

  for (int c = 0; c < tensor_scale.extent()[1]; c++) {
    for (int r = 0; r < tensor_scale.extent()[0]; ++r) {
      int f = (((c * v + r + v / 3 ) % 63) + 1);
      v += 41;
      int m = (c * v + r + v / 8 ) % 4;
      tensor_scale.at({r, c}) = ElementQScale(static_cast<float>(f) / static_cast<float>(1 << (2 + m)));
      tensor_scale1.at({r, c}) = tensor_scale.at({r, c});
    }
  }

  if (has_offsets){
    for (int c = 0; c < tensor_offset.extent()[1]; c++) {
      for (int r = 0; r < tensor_offset.extent()[0]; r += 2) {
        v = (v + 5) % 16;
        uint8_t v0 = static_cast<uint8_t>(v);
        tensor_offset.at({r, c}) = ElementQOffset(v0);
        uint8_t v1 = 0;
        if (r + 1 < tensor_offset.extent()[0]) {
          v = (v + 5) % 16;
          v1 = static_cast<uint8_t>(v);
          tensor_offset.at({r + 1, c}) = ElementQOffset(v1);
        }
        tensor_offset1.at({r/2, c}) = ElementW((v1 << 4) | v0);
      }
    }
  }

#if 0
  // Fill tensor_weight with the patterned data, so that we can use
  // print to make sure the layout matches after loaded to registers
  // in the kernel to debug errors
  int loop_val = 0;
  int offset = 3;
  for (int col_tile = 0; col_tile < tensor_weight.extent().column()/8; ++col_tile) {
    for (int row_tile = 0; row_tile < tensor_weight.extent().row()/4; ++row_tile) {
      for (int col = 0; col < 8; ++col) {
        for (int row = 0; row < 4; ++row) {
          auto weight_cord = cutlass::make_Coord(row_tile * 4 + row, col_tile * 8 + col);
          auto val = (loop_val + offset) % 256;
          tensor_weight.at(weight_cord) = ElementW(val);
          loop_val++;
          if (loop_val == 256) {
            loop_val = 0;
            offset += 11;
          }
        }
      }
    }
  }
  for (int col = 0; col < tensor_scale.extent().column(); ++col){
    int c =  col * QuantBlocking::kColumn;
    for (int row = 0; row < tensor_scale.extent().row(); ++row){
      int r = row * QuantBlocking::kRow;
      auto weight_cord = cutlass::make_Coord(r/2, c);
      int w = 0;
      if (r % 2 == 0) {
        w = int(tensor_weight.at(weight_cord) & 0x0f);
      } else {
        w = int(tensor_weight.at(weight_cord) >> 4);
      }
      tensor_scale.at({row, col}) = w;
      if (has_offsets)
        tensor_offset.at({row, col}) = ElementQOffset(w);
    }
  }
#endif

  // Prepacking weight matrix and quantization meta data ...

  PackedBuf<block_size, column_wise_blocking, has_offsets> packed_buf(k, n);
  packed_buf.pack_weights(gsl::make_span<uint8_t const>(reinterpret_cast<uint8_t const *>(tensor_weight.host_data()), tensor_weight.size()));
  packed_buf.pack_scales(gsl::make_span<MLFloat16 const>(reinterpret_cast<MLFloat16 const *>(tensor_scale1.host_data()), tensor_scale1.size()));
  if constexpr (has_offsets) {
    packed_buf.pack_offsets(gsl::make_span<uint8_t const>(reinterpret_cast<uint8_t const *>(tensor_offset1.host_data()), tensor_offset1.size()));
  }

#if 0
  // Debug verify the prepacking goes as expected. This is only for
  // debugging kernel errors.

  cutlass::HostTensor<ElementW, LayoutInputWPack> tensor_weight_prepacked(
    cutlass::make_Coord(problem_size.k(), problem_size.n()/2));
  prepack_weights_ref(problem_size.k(), problem_size.n(),
                      make_ConstMatrixRef(tensor_weight),
                      make_MatrixRef(tensor_weight_prepacked));
  const MatrixRef<uint8_t, ColumnMajorLayout>
        ref_wp1(packed_buf.GetPackedWeights(), make_Position(problem_size.k(), problem_size.n()/2));
  for (int col = 0; col < tensor_weight_prepacked.extent().column(); ++col){
    for (int row = 0; row < tensor_weight_prepacked.extent().row(); ++row){
      if (tensor_weight_prepacked.at({row, col}) != ref_wp1.at(row, col)) {
        ORT_THROW("Weight prepacking failed at row: ", row, ", col: ", col);
      }
    }
  }

  cutlass::HostTensor<ElementQScale, LayoutInputQScale> tensor_scale_prepacked(
      {problem_size.k()/QuantBlocking::kRow, problem_size.n()/QuantBlocking::kColumn});
  cutlass::HostTensor<ElementQOffset, LayoutInputQScale> tensor_offset_prepacked(
      {problem_size.k()/QuantBlocking::kRow, problem_size.n()/QuantBlocking::kColumn});

  auto scale_ref = make_ConstMatrixRef(tensor_scale);
  prepack_quant_scales_ref<ElementQScale, typename decltype(scale_ref)::Layout, QuantBlocking>(
      problem_size.k(), problem_size.n(), scale_ref,
      make_MatrixRef(tensor_scale_prepacked));
  const MatrixRef<MLFloat16, typename decltype(scale_ref)::Layout>
        ref_sp1(packed_buf.GetPackedScales(), make_Position(problem_size.k()/QuantBlocking::kRow, problem_size.n()/QuantBlocking::kColumn));
  for (int col = 0; col < tensor_scale_prepacked.extent().column(); ++col){
    for (int row = 0; row < tensor_scale_prepacked.extent().row(); ++row){
      if (tensor_scale_prepacked.at({row, col}) != ref_sp1.at(row, col)) {
        ORT_THROW("Scale prepacking failed at row: ", row, ", col: ", col);
      }
    }
  }
  if constexpr (has_offsets) {
    auto offset_ref = make_ConstMatrixRef(tensor_offset);
    prepack_quant_offsets_ref<typename decltype(offset_ref)::Layout, QuantBlocking>(
        problem_size.k(), problem_size.n(), offset_ref,
        make_MatrixRef(tensor_offset_prepacked));
    const MatrixRef<uint8_t, typename decltype(offset_ref)::Layout>
          ref_op1(packed_buf.GetPackedOffsets(), make_Position(problem_size.k()/QuantBlocking::kRow, problem_size.n()/QuantBlocking::kColumn));
    for (int col = 0; col < tensor_offset_prepacked.extent().column(); ++col){
      for (int row = 0; row < tensor_offset_prepacked.extent().row(); ++row){
        if (tensor_offset_prepacked.at({row, col}) != ref_op1.at(row, col)) {
          ORT_THROW("Offset prepacking failed at row: ", row, ", col: ", col);
        }
      }
    }
  }
#endif

  // Copy data from host to GPU...
  tensor_a.sync_device();
  tensor_d.sync_device();
  packed_buf.copy_to_gpu();

  // Construct events
  hipEvent_t finish_gemm_event;
  auto cuda_err = hipEventCreate(&finish_gemm_event);
  ORT_ENFORCE(cuda_err == hipSuccess, "Failed to create CUDA event.");

  // Run the GEMM kernel
  gsl::span<half const> a_span(reinterpret_cast<half const *>(tensor_a.device_data()), tensor_a.size());
  gsl::span<half> output_span(reinterpret_cast<half *>(tensor_d.device_data()), tensor_d.size());
  auto s = blkq4_fp16_gemm_sm80_dispatch(
    block_size, column_wise_blocking, m, n, k, nullptr,
    a_span, packed_buf.GetPackedWeightsGpu(), packed_buf.GetPackedScalesGpu(),
    (has_offsets ? packed_buf.GetPackedOffsetsGpu() : gsl::span<uint8_t const,0>{}), output_span);
  ORT_ENFORCE(s.IsOK(), s.ErrorMessage());

  // Record an event when the GEMMs are complete
  cuda_err = hipEventRecord(finish_gemm_event);
  ORT_ENFORCE(cuda_err == hipSuccess, "Failed to record CUDA event: ", hipGetErrorString(cuda_err));

  // Wait for work on the device to complete.
  cuda_err = hipEventSynchronize(finish_gemm_event);
  ORT_ENFORCE(cuda_err == hipSuccess, "Failure during sync CUDA event: ", hipGetErrorString(cuda_err));

  hipEventDestroy(finish_gemm_event);

  // Preparing reference kernel arguments
  // Dequantizing weights and running reference kernel

  using ElementInputB = ElementInputA;
  using LayoutInputB = cutlass::layout::ColumnMajor;
  cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b(
      problem_size.kn());  // <- Create dequantized matrix B with dimensions K x N
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_ref_d(
      problem_size.mn());  // <- Create matrix D with dimensions M x N used to store output from
                           // reference kernel

  // Dequantize weights and save into matrix B for reference
  for (int col = 0; col < tensor_b.extent().column(); ++col){
    for (int row = 0; row < tensor_b.extent().row(); ++row) {
      auto weight_cord = cutlass::make_Coord(row/2, col);
      auto scale_cord = cutlass::make_Coord(row / QuantBlocking::kRow, col / QuantBlocking::kColumn);
      const uint8_t offset = has_offsets ? tensor_offset.at(scale_cord) : 8;
      int w = 0;
      if (row % 2 == 0) {
        w = int(tensor_weight.at(weight_cord) & 0x0f) - offset;
      } else {
        w = int(tensor_weight.at(weight_cord) >> 4) - offset;
      }
      auto scale = tensor_scale.at(scale_cord);
      tensor_b.at({row, col}) = scale * float(w);
    }
  }
  cutlass::reference::host::TensorFill(
      tensor_ref_d.host_view());  // <- fill matrix D for reference on host with zeros

  tensor_b.sync_device();
  tensor_ref_d.sync_device();

  // Initialize alpha and beta for dot product computation
  ElementComputeEpilogue alpha = ElementComputeEpilogue(1);
  ElementComputeEpilogue beta = ElementComputeEpilogue(0);

  compute_gemm_ref<ElementInputA, LayoutInputA,
               ElementInputB, LayoutInputB,
               ElementOutput, LayoutOutput,
               ElementComputeEpilogue, ElementAccumulator>(
      problem_size,
      alpha,
      tensor_a.device_ref(),
      tensor_b.device_ref(),
      beta,
      tensor_ref_d.device_ref());

  // Wait for kernels to finish
  hipDeviceSynchronize();

  // Copy output data from CUTLASS and reference kernel to host for comparison
  tensor_d.sync_host();
  tensor_ref_d.sync_host();

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  bool passed = cutlass::reference::host::TensorEquals(
    tensor_d.host_view(),
    tensor_ref_d.host_view());
  ORT_ENFORCE(passed, "Gemm kernel result wrong!");
}

template void run_blkq4_gemm<16, true, false, true>(int m, int n, int k);
template void run_blkq4_gemm<16, true, false, false>(int m, int n, int k);
template void run_blkq4_gemm<32, true, false, true>(int m, int n, int k);
template void run_blkq4_gemm<32, true, false, false>(int m, int n, int k);
template void run_blkq4_gemm<64, true, false, true>(int m, int n, int k);
template void run_blkq4_gemm<64, true, false, false>(int m, int n, int k);
template void run_blkq4_gemm<16, false, false, true>(int m, int n, int k);
template void run_blkq4_gemm<16, false, false, false>(int m, int n, int k);
template void run_blkq4_gemm<32, false, false, true>(int m, int n, int k);
template void run_blkq4_gemm<32, false, false, false>(int m, int n, int k);
template void run_blkq4_gemm<64, false, false, true>(int m, int n, int k);
template void run_blkq4_gemm<64, false, false, false>(int m, int n, int k);
template void run_blkq4_gemm<16, true, true, true>(int m, int n, int k);
template void run_blkq4_gemm<16, true, true, false>(int m, int n, int k);
template void run_blkq4_gemm<32, true, true, true>(int m, int n, int k);
template void run_blkq4_gemm<32, true, true, false>(int m, int n, int k);
template void run_blkq4_gemm<64, true, true, true>(int m, int n, int k);
template void run_blkq4_gemm<64, true, true, false>(int m, int n, int k);
template void run_blkq4_gemm<16, false, true, true>(int m, int n, int k);
template void run_blkq4_gemm<16, false, true, false>(int m, int n, int k);
template void run_blkq4_gemm<32, false, true, true>(int m, int n, int k);
template void run_blkq4_gemm<32, false, true, false>(int m, int n, int k);
template void run_blkq4_gemm<64, false, true, true>(int m, int n, int k);
template void run_blkq4_gemm<64, false, true, false>(int m, int n, int k);

}  // namespace test
}  // namespace cuda
}  // namespace onnxruntime
